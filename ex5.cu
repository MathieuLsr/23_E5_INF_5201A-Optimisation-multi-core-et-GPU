
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 1000

struct hipComplex {
    float r;
    float i;

    __device__ hipComplex( float a, float b ) : r(a), i(b) {}

    __device__ float magnitude2( void ) {
        return r * r + i * i;
    }
    
    __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    
    __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
};

__device__ int julia( int x, int y ) {
    
    const float scale = 0.5;
    float jx = scale * (float)(N/2 - x)/(N/2);
    float jy = scale * (float)(N/2 - y)/(N/2);
    
    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);
    
    int i = 0;
    for (i=0; i<200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000) return 0;
    }
    
    return 1;
}

//#define THREADS 

__global__ void kernel( unsigned char *ptr ) {

    #ifdef THREADS
    int x = threadIdx.x ;
    int y = threadIdx.y ; 
    int offset = x + y * blockDim.x;
    #else 
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;
    #endif

    int juliaValue = julia( x, y );
    ptr[offset] = 255 * juliaValue;
    
}

int main() {

    printf("==== Start... ====\n\n") ; 

    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );

    unsigned char tab[N*N];
    unsigned char *gpu_tab ;

    

    for(int i=0 ; i < N*N ; i++){
        tab[i] = 0 ; 
    }
        


    if(hipMalloc( (void**)&gpu_tab, N*N* sizeof(unsigned char) ) != hipSuccess){
        printf("Erreur hipMalloc #1\n") ; 
        return -1 ; 
    }
    
    if(hipMemcpy( gpu_tab, tab, N*N * sizeof(unsigned char), hipMemcpyHostToDevice )!= hipSuccess){
        printf("Erreur hipMemcpy #1\n") ; 
        return -1 ; 
    }


    dim3 grille(N,N) ; 

    hipEventRecord( start, 0 );
    kernel<<<grille, 1>>>(gpu_tab) ; 
    hipEventRecord( stop, 0 ) ;
    hipEventSynchronize( stop ) ;

    if(hipMemcpy( tab, gpu_tab, N*N * sizeof(unsigned char),hipMemcpyDeviceToHost )!= hipSuccess){
        printf("Erreur hipMemcpy #2\n") ; 
        return -1 ; 
    }

    FILE* f=fopen("julia.pgm","wb");
    fprintf(f,"P5\n %d %d\n255\n",N,N);
    for (int i=0; i< N*N; i++){
        fputc(tab[i],f);
    }
    fclose(f);

    float elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop );
    printf( "Time to generate: %3.1f ms\n", elapsedTime );
    hipEventDestroy( start );
    hipEventDestroy( stop );

    printf("\n") ; 
    printf("==== END ====\n\n") ; 

}

/*

<<<1,1>>> : 1.5 ms 
<<<1000,1>>> : 0.2 ms
<<<1,1000>>> : 0.1 ms

*/